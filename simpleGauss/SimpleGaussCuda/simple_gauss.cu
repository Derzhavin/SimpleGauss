#include "hip/hip_runtime.h"
//
// Created by denis on 09.12.2021.
//

#include "simple_gauss.cuh"

//template<typename T>
//__global__ void simple_gauss_kernel (T *g_mat, T *g_sol_mat, size_t rows, size_t cols)
//{
//
//}
//
//template<typename T>
//void simple_gauss(dim3 grid, dim3 block, T *g_mat, T *g_sol_mat, size_t rows, size_t cols)
//{
//    simple_gauss_kernel<T><<<grid, block>>>(g_mat, g_sol_mat, rows, cols);
//}

__global__ void simple_gauss_kernel (double  *g_mat, double  *g_sol_mat, unsigned int rows, unsigned int cols)
{
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int k = 0; k < rows - 1; ++k)
    {
        if ( i > k)
        {
            double pivot = g_mat[k * cols + k];
            double coeff = g_mat[i * cols + k];
            coeff /= pivot;
            double mat_kj = g_mat[k * cols + j];
            g_mat[i * cols + j] -= coeff * mat_kj;
        }
    }
}

void simple_gauss(dim3 grid, dim3 block, double *g_mat, double *g_sol_mat, size_t rows, size_t cols)
{
    simple_gauss_kernel<<<grid, block>>>(g_mat, g_sol_mat, rows, cols);
}