#include "hip/hip_runtime.h"
//
// Created by denis on 09.12.2021.
//

#include "simple_gauss.cuh"
//template<typename T>
//__global__ void simple_gauss_kernel (T *g_mat, T *g_sol_mat, size_t rows, size_t cols)
//{
//
//}
//
//template<typename T>
//void simple_gauss(dim3 grid, dim3 block, T *g_mat, T *g_sol_mat, size_t rows, size_t cols)
//{
//    simple_gauss_kernel<T><<<grid, block>>>(g_mat, g_sol_mat, rows, cols);
//}
//
#include <cstdio>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

__global__ void simple_gauss_kernel (float  *g_mat, float  *g_mat_sol, unsigned int rows, unsigned int cols)

//__global__ void simple_gauss_kernel (double  *g_mat, double  *g_mat_sol, unsigned int rows, unsigned int cols)
{
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k;

    // Прямой проход
    for (k = 0; k < rows - 1; ++k)
    {
        if ( i > k && j < cols)
        {
            float pivot = g_mat[k * cols + k];
            float coeff = g_mat[i * cols + k];
            coeff /= pivot;
            float mat_kj = g_mat[k * cols + j];
            g_mat[i * cols + j] -= coeff * mat_kj;
        }
        __syncthreads();
    }

    __syncthreads();

    // Обратный проход
    for (k = 0; k < rows; ++k)
    {
        if (i == rows - 1 - k && j == 0)
        {
            g_mat_sol[rows - 1 - k] = g_mat[(rows - 1 - k) * cols + rows];
        }
        __syncthreads();

        if (i == k && j >= rows - 1 - k && j < rows - 1)
        {
             atomicAdd(&g_mat_sol[rows - 1 - k], -(g_mat[(rows - 1 - k) * cols + j + 1] * g_mat_sol[j + 1]));
        }
        __syncthreads();

       if (i == rows - 1 - k && j == 0)
       {
           g_mat_sol[rows - 1 - k] /= g_mat[(rows - 1 - k) * cols + rows - 1 - k];
       }
        __syncthreads();
    }
}
void simple_gauss(dim3 grid, dim3 block, float *g_mat, float *g_sol_mat, size_t rows, size_t cols)

//void simple_gauss(dim3 grid, dim3 block, double *g_mat, double *g_sol_mat, size_t rows, size_t cols)
{
//    hipDeviceSynchronize();
    simple_gauss_kernel<<<grid, block>>>(g_mat, g_sol_mat, rows, cols);
    CHECK(hipDeviceSynchronize());
}