#include "hip/hip_runtime.h"
//
// Created by denis on 09.12.2021.
//

#include "simple_gauss.cuh"

#include <cstdio>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

__global__ void simple_gauss_straight_passage_kernel (double  *g_mat, unsigned int n, unsigned int number_row, unsigned int number_col)

//__global__ void simple_gauss_kernel (double  *g_mat, double  *g_mat_sol, unsigned int rows, unsigned int cols)
{
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i > number_row && j < n + 1)
    {
        double pivot = g_mat[number_row * (n  + 1) + number_row];
        double coeff = g_mat[i * (n  + 1) + number_row];
        coeff /= pivot;
        double mat_kj = g_mat[number_row * (n  + 1) + j];
        double res = coeff * mat_kj;
        g_mat[i * (n  + 1) + j] -= res;
    }
}

//__global__ void simple_gauss_backward_passage_kernel(double  *g_mat, double  *g_mat_sol, unsigned int n, unsigned int number_row, unsigned int number_col)
//{
//    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
//    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (i == n - 1 - number_row  && j >= number_row && j < n - 1)
//    {
//        atomicAdd(&g_mat_sol[number_row], -(g_mat[number_row * (n + 1) + j + 1] * g_mat_sol[j + 1]));
//    }
//    __syncthreads();
//
//    if (i == number_row && j == 0)
//    {
//        g_mat_sol[number_row] += g_mat[number_row * (n + 1) + n];
//        g_mat_sol[number_row] /= g_mat[number_row * (n + 1) + number_row];
//    }
//}
void simple_gauss(double *h_mat, double *h_mat_sol, unsigned int n)
{
    double *d_mat;
    hipMalloc((void **) &d_mat, n * (n + 1) * sizeof(double));
    hipMemcpy(d_mat, h_mat, n * (n + 1) * sizeof(double), hipMemcpyHostToDevice);

//    double *d_mat_sol;
//    hipMalloc((void **) &d_mat_sol, n * sizeof(double));
//    hipMemcpy(d_mat, h_mat, n * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlockDim = 16;
    dim3 block(threadsPerBlockDim, threadsPerBlockDim, 1);
    int blocksPerGridDimX = ceilf((n + 1) / (double)threadsPerBlockDim);
    int blocksPerGridDimY = ceilf(n / (double)threadsPerBlockDim);
//    printf("%d, %d\n", blocksPerGridDimY, blocksPerGridDimX);
    dim3 grid(blocksPerGridDimX, blocksPerGridDimY, 1);

    for (int i = 0; i < n - 1; ++i)
    {
        simple_gauss_straight_passage_kernel<<<grid, block>>>(d_mat, n, i, i);
//        CHECK(hipDeviceSynchronize());
        CHECK(hipStreamSynchronize(0));
        CHECK(hipGetLastError());
    }
//    for (int i = 0; i < n; ++i)
//    {
//        simple_gauss_backward_passage_kernel<<<grid, block>>>(d_mat, d_mat_sol, n, n - 1 - i, n - 1 - i);
//        CHECK(hipDeviceSynchronize());
//        CHECK(hipGetLastError());
//    }
//    hipMemcpy(h_mat_sol, d_mat_sol, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_mat, d_mat, n * (n + 1) * sizeof(double), hipMemcpyDeviceToHost);

    for (size_t k = 0, i; k < n; ++k)
    {
        h_mat_sol[n - 1 - k] = h_mat[(n - 1 - k) * (n + 1) + n];
        for (i = n - 1 - k; i < n - 1; ++i)
            h_mat_sol[n - 1 - k] -= h_mat[(n - 1 - k) * (n + 1) + i + 1] * h_mat_sol[i + 1];
        h_mat_sol[n - 1 - k] /= h_mat[(n - 1 - k) * (n + 1) + n - 1 - k];
    }

    hipFree(d_mat);
//    hipFree(d_mat_sol);
}