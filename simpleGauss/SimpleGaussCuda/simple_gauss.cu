#include "hip/hip_runtime.h"
//
// Created by denis on 09.12.2021.
//

#include "simple_gauss.cuh"

//template<typename T>
//__global__ void simple_gauss (T *g_mat, T *g_sol_mat, size_t rows, size_t cols)
//{
//
//}
//
//template<typename T>
//void simple_gauss(dim3 grid, dim3 block, T *g_mat, T *g_sol_mat, size_t rows, size_t cols)
//{
//    simple_gauss<T><<<grid, block>>>(g_mat, g_sol_mat, rows, cols);
//}

__global__ void simple_gauss (double  *g_mat, double  *g_sol_mat, size_t rows, size_t cols)
{

}

void simple_gauss(dim3 grid, dim3 block, double *g_mat, double *g_sol_mat, size_t rows, size_t cols)
{
    simple_gauss<<<grid, block>>>(g_mat, g_sol_mat, rows, cols);
}